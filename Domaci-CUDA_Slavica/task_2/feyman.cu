
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include<sys/time.h>

#define NUMBER_OF_THREADS_PER_BLOCK 1024

double wall_time(void) {
    struct timeval tv;

    gettimeofday(&tv,NULL);

    return (double)tv.tv_usec / 1000000 + (double)tv.tv_sec;
}

__device__ double pow_device(double x, int degree) {
  if(degree == 2) return x * x;
}

int i4_ceiling(double x)
{
  int value = (int)x;
  if (value < x)
    value = value + 1;
  return value;
}

int i4_min(int i1, int i2)
{
  int value;
  if (i1 < i2)
    value = i1;
  else
    value = i2;
  return value;
}
__device__ double potential(double a, double b, double c, double x, double y, double z)
{
  return 2.0 * (pow_device(x / a / a, 2) + pow_device(y / b / b, 2) + pow_device(z / c / c, 2)) + 1.0 / a / a + 1.0 / b / b + 1.0 / c / c;
}

__device__ double r8_uniform_01(int *seed)
{
  int k;
  double r;

  k = *seed / 127773;

  *seed = 16807 * (*seed - k * 127773) - k * 2836;

  if (*seed < 0)
  {
    *seed = *seed + 2147483647;
  }
  r = (double)(*seed) * 4.656612875E-10;

  return r;
}

void timestamp(void)
{
#define TIME_SIZE 40

  static char time_buffer[TIME_SIZE];
  const struct tm *tm;
  time_t now;

  now = time(NULL);
  tm = localtime(&now);

  strftime(time_buffer, TIME_SIZE, "%d %B %Y %I:%M:%S %p", tm);

  printf("%s\n", time_buffer);

  return;
#undef TIME_SIZE
}

__global__ void feyman_kernel(int N, int ni, int nj, int nk, double a, double b, double c, double* err, int* n_inside) {

  __shared__ double wt;

  int i = blockIdx.x + 1;
  int j = blockIdx.y + 1;
  int k = blockIdx.z + 1;

  double x = ((double)(ni - i) * (-a) + (double)(i - 1) * a) / (double)(ni - 1);
  double y = ((double)(nj - j) * (-b) + (double)(j - 1) * b) / (double)(nj - 1);
  double z = ((double)(nk - k) * (-c) + (double)(k - 1) * c) / (double)(nk - 1);

  double chk;
  double dx;
  double dy;
  double dz;
  const double h = 0.001;
  double stepsz;
  int seed = 123456789 + threadIdx.x;
  int steps;
  int trial;
  double us;
  double ut;
  double vh;
  double vs;
  double x1;
  double x2;
  double x3;
  double w;
  double w_exact;
  double we;
  steps = 0;

  const int dim = 3;
  stepsz = sqrt((double)dim * h);

  chk = pow_device(x / a, 2) + pow_device(y / b, 2) + pow_device(z / c, 2);
  
  if (1.0 < chk)
  {
    return;
  }
  
  // Na nivou bloka, inkrementujemo n_inside
  if (threadIdx.x == 0) {
    atomicAdd(n_inside, 1);
    wt = 0.0;
  }
  
  w_exact = exp(pow_device(x / a, 2) + pow_device(y / b, 2) + pow_device(z / c, 2) - 1.0);

  __syncthreads();

  double mywt = 0;
  for (trial = threadIdx.x; trial < N; trial += NUMBER_OF_THREADS_PER_BLOCK)
  {
    x1 = x;
    x2 = y;
    x3 = z;
    w = 1.0;
    chk = 0.0;
    while (chk < 1.0)
    {
      ut = r8_uniform_01(&seed);
      if (ut < 1.0 / 3.0)
      {
        us = r8_uniform_01(&seed) - 0.5;
        if (us < 0.0)
          dx = -stepsz;
        else
          dx = stepsz;
      }
      else
        dx = 0.0;

      ut = r8_uniform_01(&seed);
      if (ut < 1.0 / 3.0)
      {
        us = r8_uniform_01(&seed) - 0.5;
        if (us < 0.0)
          dy = -stepsz;
        else
          dy = stepsz;
      }
      else
        dy = 0.0;

      ut = r8_uniform_01(&seed);
      if (ut < 1.0 / 3.0)
      {
        us = r8_uniform_01(&seed) - 0.5;
        if (us < 0.0)
          dz = -stepsz;
        else
          dz = stepsz;
      }
      else
        dz = 0.0;

      vs = potential(a, b, c, x1, x2, x3);
      x1 = x1 + dx;
      x2 = x2 + dy;
      x3 = x3 + dz;

      steps++;

      vh = potential(a, b, c, x1, x2, x3);

      we = (1.0 - h * vs) * w;
      w = w - 0.5 * h * (vh * we + vs * w);

      chk = pow_device(x1 / a, 2) + pow_device(x2 / b, 2) + pow_device(x3 / c, 2);
    }
    mywt += w;
  }

  atomicAdd(&wt, mywt); 
  __syncthreads();

  if (threadIdx.x == 0) {
    wt = wt/ (double)(N);
  
    atomicAdd(err, pow_device(w_exact - wt, 2));
  }
}

int main(int arc, char **argv)
{
  double a = 3.0;
  double b = 2.0;
  double c = 1.0;
  int dim = 3;
  double err;
  double h = 0.001;
  int n_inside;
  int ni;
  int nj;
  int nk;
  double stepsz;

  double w_time;

  int N = atoi(argv[1]);
  timestamp();
  w_time = wall_time();

  printf("A = %f\n", a);
  printf("B = %f\n", b);
  printf("C = %f\n", c);
  printf("N = %d\n", N);
  printf("H = %6.4f\n", h);

  stepsz = sqrt((double)dim * h);

  if (a == i4_min(i4_min(a, b), c))
  {
    ni = 6;
    nj = 1 + i4_ceiling(b / a) * (ni - 1);
    nk = 1 + i4_ceiling(c / a) * (ni - 1);
  }
  else if (b == i4_min(i4_min(a, b), c))
  {
    nj = 6;
    ni = 1 + i4_ceiling(a / b) * (nj - 1);
    nk = 1 + i4_ceiling(c / b) * (nj - 1);
  }
  else
  {
    nk = 6;
    ni = 1 + i4_ceiling(a / c) * (nk - 1);
    nj = 1 + i4_ceiling(b / c) * (nk - 1);
  }

  err = 0.0;
  n_inside = 0;

  dim3 dimGrid(ni, nj, nk);
  dim3 dimBlock(NUMBER_OF_THREADS_PER_BLOCK);

  int* device_n_inside;
  double* device_err;

  hipMalloc((void**)&device_n_inside, sizeof(int));
  hipMalloc((void**)&device_err, sizeof(double));

  hipMemcpy(device_n_inside, &n_inside, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_err, &err, sizeof(double), hipMemcpyHostToDevice);

  feyman_kernel<<< dimGrid, dimBlock >>>(N, ni, nj, nk, a, b, c, device_err, device_n_inside); 
  hipDeviceSynchronize();

  hipMemcpy(&n_inside, device_n_inside, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&err, device_err, sizeof(double), hipMemcpyDeviceToHost);

  hipFree(device_n_inside);
  hipFree(device_err);

  err = sqrt(err / (double)(n_inside));

  printf("\n\nRMS absolute error in solution = %e\n", err);
  w_time = wall_time() - w_time;
  timestamp();

  FILE *fpt;
  fpt = fopen("task_2.csv", "a");
  fprintf(fpt,"CUDA, %d, %e, %f\n", N, err, w_time);
  fclose(fpt);

  return 0;
}
