#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define NUMBER_OF_THREADS_PER_BLOCK 1024

__device__ double powCuda(double x, int) {
  return x * x;
}

int i4_ceiling(double x)
{
  int value = (int)x;
  if (value < x)
    value = value + 1;
  return value;
}

int i4_min(int i1, int i2)
{
  int value;
  if (i1 < i2)
    value = i1;
  else
    value = i2;
  return value;
}
__device__
double potential(double a, double b, double c, double x, double y, double z)
{
  return 2.0 * (powCuda(x / a / a, 2) + powCuda(y / b / b, 2) + powCuda(z / c / c, 2)) + 1.0 / a / a + 1.0 / b / b + 1.0 / c / c;
}

__device__
double r8_uniform_01(int *seed)
{
  int k;
  double r;

  k = *seed / 127773;

  *seed = 16807 * (*seed - k * 127773) - k * 2836;

  if (*seed < 0)
  {
    *seed = *seed + 2147483647;
  }
  r = (double)(*seed) * 4.656612875E-10;

  return r;
}

void timestamp(void)
{
#define TIME_SIZE 40

  static char time_buffer[TIME_SIZE];
  const struct tm *tm;
  time_t now;

  now = time(NULL);
  tm = localtime(&now);

  strftime(time_buffer, TIME_SIZE, "%d %B %Y %I:%M:%S %p", tm);

  printf("%s\n", time_buffer);

  return;
#undef TIME_SIZE
}



__global__ void feymanKernel(int N, int ni, int nj, int nk, double* err, int* n_inside, double* wt) {

  int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;

  const double a = 3.0;
  const double b = 2.0;
  const double c = 1.0;
  int i = blockIdx.x + 1;
  int j = blockIdx.y + 1;
  int k = blockIdx.z + 1;

  double x = ((double)(ni - i) * (-a) + (double)(i - 1) * a) / (double)(ni - 1);
  double y = ((double)(nj - j) * (-b) + (double)(j - 1) * b) / (double)(nj - 1);
  double z = ((double)(nk - k) * (-c) + (double)(k - 1) * c) / (double)(nk - 1);

  double chk;
  double dx;
  double dy;
  double dz;
  const double h = 0.001;
  double stepsz;
  int seed = 123456789 + threadIdx.x;
  int steps;
  int trial;
  double us;
  double ut;
  double vh;
  double vs;
  double x1;
  double x2;
  double x3;
  double w;
  double w_exact;
  double we;
  steps = 0;

  const int dim = 3;
  stepsz = sqrt((double)dim * h);

  chk = powCuda(x / a, 2) + powCuda(y / b, 2) + powCuda(z / c, 2);
  
  if (1.0 < chk)
  {
    w_exact = 1.0;
    wt[blockId] = 1.0;
    return;
  }
  
  // Na nivou bloka, inkrementujemo n_inside
  if (threadIdx.x == 0) {
    atomicAdd(n_inside, 1);
    wt[blockId] = 0;
  }
  
  w_exact = exp(powCuda(x / a, 2) + powCuda(y / b, 2) + powCuda(z / c, 2) - 1.0);
  __syncthreads();
  double mywt = 0;
  for (trial = threadIdx.x; trial < N; trial += NUMBER_OF_THREADS_PER_BLOCK)
  {
    x1 = x;
    x2 = y;
    x3 = z;
    w = 1.0;
    chk = 0.0;
    while (chk < 1.0)
    {
      ut = r8_uniform_01(&seed);
      if (ut < 1.0 / 3.0)
      {
        us = r8_uniform_01(&seed) - 0.5;
        if (us < 0.0)
          dx = -stepsz;
        else
          dx = stepsz;
      }
      else
        dx = 0.0;

      ut = r8_uniform_01(&seed);
      if (ut < 1.0 / 3.0)
      {
        us = r8_uniform_01(&seed) - 0.5;
        if (us < 0.0)
          dy = -stepsz;
        else
          dy = stepsz;
      }
      else
        dy = 0.0;

      ut = r8_uniform_01(&seed);
      if (ut < 1.0 / 3.0)
      {
        us = r8_uniform_01(&seed) - 0.5;
        if (us < 0.0)
          dz = -stepsz;
        else
          dz = stepsz;
      }
      else
        dz = 0.0;

      vs = potential(a, b, c, x1, x2, x3);
      x1 = x1 + dx;
      x2 = x2 + dy;
      x3 = x3 + dz;

      steps++;

      vh = potential(a, b, c, x1, x2, x3);

      we = (1.0 - h * vs) * w;
      w = w - 0.5 * h * (vh * we + vs * w);

      chk = powCuda(x1 / a, 2) + powCuda(x2 / b, 2) + powCuda(x3 / c, 2);
    }
    // atomicAdd(&(wt[blockId]), w);  
    mywt += w;
  }

  atomicAdd(&(wt[blockId]), mywt); 
  __syncthreads();

  if (threadIdx.x == 0) {
    wt[blockId] = wt[blockId] / (double)(N);
  
    atomicAdd(err, powCuda(w_exact - wt[blockId], 2));
  }
}

// print na stdout upotrebiti u validaciji paralelnog resenja
int main(int arc, char **argv)
{
  double a = 3.0;
  double b = 2.0;
  double c = 1.0;
  int dim = 3;
  double err;
  double h = 0.001;
  int n_inside;
  int ni;
  int nj;
  int nk;
  double stepsz;

  int N = atoi(argv[1]);
  timestamp();

  printf("A = %f\n", a);
  printf("B = %f\n", b);
  printf("C = %f\n", c);
  printf("N = %d\n", N);
  printf("H = %6.4f\n", h);

  stepsz = sqrt((double)dim * h);

  if (a == i4_min(i4_min(a, b), c))
  {
    ni = 6;
    nj = 1 + i4_ceiling(b / a) * (ni - 1);
    nk = 1 + i4_ceiling(c / a) * (ni - 1);
  }
  else if (b == i4_min(i4_min(a, b), c))
  {
    nj = 6;
    ni = 1 + i4_ceiling(a / b) * (nj - 1);
    nk = 1 + i4_ceiling(c / b) * (nj - 1);
  }
  else
  {
    nk = 6;
    ni = 1 + i4_ceiling(a / c) * (nk - 1);
    nj = 1 + i4_ceiling(b / c) * (nk - 1);
  }

  err = 0.0;
  n_inside = 0;

  dim3 dimGrid(ni, nj, nk);
  dim3 dimBlock(NUMBER_OF_THREADS_PER_BLOCK);

  int* device_n_inside;
  double* device_err;
  double* wt_device;

  int sharedMemSize = NUMBER_OF_THREADS_PER_BLOCK * sizeof(double);

  hipMalloc((void**)&device_n_inside, sizeof(int));
  hipMalloc((void**)&device_err, sizeof(double));
  hipMalloc((void**)&wt_device, ni * nj * nk * sizeof(double));

  hipMemcpy(device_n_inside, &n_inside, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_err, &err, sizeof(double), hipMemcpyHostToDevice);

  // int N, double ni, double nj, double nk, double* err, int* n_inside

  feymanKernel<<< dimGrid, dimBlock >>>(N, ni, nj, nk, device_err, device_n_inside, wt_device); 
  hipDeviceSynchronize();

  hipMemcpy(&n_inside, device_n_inside, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&err, device_err, sizeof(double), hipMemcpyDeviceToHost);

  hipFree(device_n_inside);
  hipFree(device_err);
  hipFree(wt_device);
  
  err = sqrt(err / (double)(n_inside));

  printf("\n\nRMS absolute error in solution = %e\n", err);
  timestamp();

  return 0;
}
