#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
// #include <omp.h>

#define NUMBER_OF_THREADS_PER_BLOCK 1024

#define mm 15
#define npart 4 * mm *mm *mm
/*
 *  Function declarations
 */

void dfill(int, double, double[], int);

void domove(int, double[], double[], double[], double);

void dscal(int, double, double[], int);

void fcc(double[], int, int, double);

__global__ void forces(int, double*, double*, double, double, double*, double*);

double
mkekin(int, double[], double[], double, double);

void mxwell(double[], int, double, double);

void prnout(int, double, double, double, double, double, double, int, double);

double
velavg(int, double[], double, double);

double
secnds(void);

/*
 *  Variable declarations
 */

double epot;
double vir;
double count;

/*
 *  Main program : Molecular Dynamics simulation.
 */
int main()
{
  int move;
  double x[npart * 3], vh[npart * 3], f[npart * 3];
  double ekin;
  double vel;
  double sc;
  double start, time;

  /*
   *  Parameter definitions
   */

  double den = 0.83134;
  double side = pow((double)npart / den, 0.3333333);
  double tref = 0.722;
  double rcoff = (double)mm / 4.0;
  double h = 0.064;
  int irep = 10;
  int istop = 20;
  int iprint = 5;
  int movemx = 20;

  double a = side / (double)mm;
  double hsq = h * h;
  double hsq2 = hsq * 0.5;
  double tscale = 16.0 / ((double)npart - 1.0);
  double vaver = 1.13 * sqrt(tref / 24.0);

  /*
   *  Initial output
   */

  printf(" Molecular Dynamics Simulation example program\n");
  printf(" ---------------------------------------------\n");
  printf(" number of particles is ............ %6d\n", npart);
  printf(" side length of the box is ......... %13.6f\n", side);
  printf(" cut off is ........................ %13.6f\n", rcoff);
  printf(" reduced temperature is ............ %13.6f\n", tref);
  printf(" basic timestep is ................. %13.6f\n", h);
  printf(" temperature scale interval ........ %6d\n", irep);
  printf(" stop scaling at move .............. %6d\n", istop);
  printf(" print interval .................... %6d\n", iprint);
  printf(" total no. of steps ................ %6d\n", movemx);

  /*
   *  Generate fcc lattice for atoms inside box
   */
  fcc(x, npart, mm, a);
  /*
   *  Initialise velocities and forces (which are zero in fcc positions)
   */
  mxwell(vh, 3 * npart, h, tref);
  dfill(3 * npart, 0.0, f, 1);
  /*
   *  Start of md
   */
  printf("\n    i       ke         pe            e         temp   "
         "   pres      vel      rp\n  -----  ----------  ----------"
         "  ----------  --------  --------  --------  ----\n");

  start = secnds();

  for (move = 1; move <= movemx; move++)
  {

    /*
     *  Move the particles and partially update velocities
     */
    domove(3 * npart, x, vh, f, side);

    /*
     *  Compute forces in the new positions and accumulate the virial
     *  and potential energy.
     */

    int blockSize = (npart - 1 + NUMBER_OF_THREADS_PER_BLOCK)/NUMBER_OF_THREADS_PER_BLOCK;

    dim3 dimGrid(blockSize);
    dim3 dimBlock(NUMBER_OF_THREADS_PER_BLOCK);

    int sharedMemSize = NUMBER_OF_THREADS_PER_BLOCK * npart * sizeof(double);
    double* f_device;
    double* x_device;
    double* epot_device;
    double* vir_device;
    vir = 0.0;
    epot = 0.0;
    hipMalloc((void**)&f_device,npart * 3 * sizeof(double));
    hipMalloc((void**)&x_device,npart * 3 * sizeof(double));
    hipMalloc((void**)&epot_device,sizeof(double));
    hipMalloc((void**)&vir_device,sizeof(double));

    double npart_res = 4 * mm *mm *mm;

    hipMemcpy(f_device, f, npart * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(x_device, x, npart * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(epot_device, &epot, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vir_device, &vir, sizeof(double), hipMemcpyHostToDevice);

      //(int *npart, double *x, double *f, double *side, double *rcoff, double *vir, double *epot)
    forces<<< dimGrid, dimBlock >>>(npart, x_device, f_device, side, rcoff, vir_device, epot_device);

    hipDeviceSynchronize();

    hipMemcpy(f, f_device, npart * 3 * sizeof(double), hipMemcpyDeviceToHost);

    hipMemcpy(x, x_device, npart * 3 * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&epot, epot_device, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&vir, vir_device, sizeof(double), hipMemcpyDeviceToHost);

    // printf("\n%lf    %lf\n", epot, vir);

    hipFree(f_device);
    hipFree(x_device);
    hipFree(epot_device);
    hipFree(vir_device);
    

    /*
     *  Scale forces, complete update of velocities and compute k.e.
     */
    ekin = mkekin(npart, f, vh, hsq2, hsq);

    /*
     *  Average the velocity and temperature scale if desired
     */
    vel = velavg(npart, vh, vaver, h);
    if (move < istop && fmod(move, irep) == 0)
    {
      sc = sqrt(tref / (tscale * ekin));
      dscal(3 * npart, sc, vh, 1);
      ekin = tref / tscale;
    }

    /*
     *  Sum to get full potential energy and virial
     */
    if (fmod(move, iprint) == 0)
      prnout(move, ekin, epot, tscale, vir, vel, count, npart, den);
  }

  time = secnds() - start;

  printf("Time =  %f\n", (float)time);
}

time_t starttime = 0;

double secnds()
{

  return time(0);//omp_get_wtime();
}
