
#include <hip/hip_runtime.h>

/*
 *  Compute forces and accumulate the virial and the potential
 */


__global__ void forces(int npart, double *x, double *f, double side, double rcoff, double *vir, double *epot)
{
  int i, j;

  i = (blockIdx.x * blockDim.x + threadIdx.x) * 3;

  double sideh, rcoffs;
  double xi, yi, zi, fxi, fyi, fzi, xx, yy, zz;
  double rd, rrd, rrd2, rrd3, rrd4, rrd6, rrd7, r148;
  double forcex, forcey, forcez;

  // vir = 0.0;
  // epot = 0.0;
  sideh = 0.5 * side;
  rcoffs = rcoff * rcoff;

  xi = x[i];
  yi = x[i + 1];
  zi = x[i + 2];
  fxi = 0.0;
  fyi = 0.0;
  fzi = 0.0;

  for (j = i + 3; j < npart * 3; j += 3)
  {
    xx = xi - x[j];
    yy = yi - x[j + 1];
    zz = zi - x[j + 2];
    if (xx < -sideh)
      xx += side;
    if (xx > sideh)
      xx -= side;
    if (yy < -sideh)
      yy += side;
    if (yy > sideh)
      yy -= side;
    if (zz < -sideh)
      zz += side;
    if (zz > sideh)
      zz -= side;
    rd = xx * xx + yy * yy + zz * zz;

    if (rd <= rcoffs)
    {
      rrd = 1.0 / rd;
      rrd2 = rrd * rrd;
      rrd3 = rrd2 * rrd;
      rrd4 = rrd2 * rrd2;
      rrd6 = rrd2 * rrd4;
      rrd7 = rrd6 * rrd;
      // epot += (rrd6 - rrd3);
      atomicAdd(epot, (rrd6 - rrd3));
      r148 = rrd7 - 0.5 * rrd4;
      // vir -= rd * r148;
      atomicAdd(vir, -(rd * r148));
      forcex = xx * r148;
      fxi += forcex;
      // f[j] -= forcex;
      atomicAdd(&(f[j]), -forcex);
      forcey = yy * r148;
      fyi += forcey;
      // f[j + 1] -= forcey;
      atomicAdd(&(f[j + 1]), -forcey);
      forcez = zz * r148;
      fzi += forcez;
      // f[j + 2] -= forcez;
      atomicAdd(&(f[j + 2]), -forcez);
    }
  }
  // f[i] += fxi;
  atomicAdd(&(f[i]), fxi);
  // f[i + 1] += fyi;
  atomicAdd(&(f[i + 1]), fyi);
  // f[i + 2] += fzi;
  atomicAdd(&(f[i + 2]), fzi);
}
