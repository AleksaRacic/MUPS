
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include<sys/time.h>

double wall_time(void) {
    struct timeval tv;

    gettimeofday(&tv,NULL);

    return (double)tv.tv_usec / 1000000 + (double)tv.tv_sec;
}

double cpu_time(void)
{
  double value;

  value = (double)clock() / (double)CLOCKS_PER_SEC;

  return value;
}

__global__ void primeNumberKernel(int n, int *total)
{
  int j;
  int number = 2 + blockIdx.x * blockDim.x + threadIdx.x;

  if (number > n)
    return;

  for (j = 2; j < number; j ++)
  {
    if ((number % j) == 0) return;
  }

  atomicAdd(total, 1);
}

#define NUMBER_OF_THREADS_PER_BLOCK 1024

int prime_number(int n)
{
  int total;
  total = 0;

  int numBlock = (n + NUMBER_OF_THREADS_PER_BLOCK - 1) / NUMBER_OF_THREADS_PER_BLOCK;

  dim3 dimGrid(numBlock);
  dim3 dimBlock(NUMBER_OF_THREADS_PER_BLOCK);

  int *total_device;
  hipMalloc((void **)&total_device, sizeof(int));
  hipMemcpy(total_device, &total, sizeof(int), hipMemcpyHostToDevice);

  primeNumberKernel<<<dimGrid, dimBlock>>>(n, total_device);

  hipDeviceSynchronize();

  hipMemcpy(&total, total_device, sizeof(int), hipMemcpyDeviceToHost);

  hipFree(total_device);

  return total;
}

void timestamp(void)
{
#define TIME_SIZE 40

  static char time_buffer[TIME_SIZE];
  const struct tm *tm;
  size_t len;
  time_t now;

  now = time(NULL);
  tm = localtime(&now);

  len = strftime(time_buffer, TIME_SIZE, "%d %B %Y %I:%M:%S %p", tm);

  printf("%s\n", time_buffer);

  return;
#undef TIME_SIZE
}

void test(int n_lo, int n_hi, int n_factor);

int main(int argc, char *argv[])
{
  int n_factor;
  int n_hi;
  int n_lo;

  double w_time;

  timestamp();
  printf("\n");
  printf("PRIME TEST\n");

  if (argc != 4)
  {
    n_lo = 1;
    n_hi = 131072;
    n_factor = 2;
  }
  else
  {
    n_lo = atoi(argv[1]);
    n_hi = atoi(argv[2]);
    n_factor = atoi(argv[3]);
  }

  w_time = wall_time();
  test(n_lo, n_hi, n_factor);
  w_time = wall_time() - w_time;
  
  printf("\n");
  printf("PRIME_TEST\n");
  printf("  Normal end of execution.\n");
  printf("\n");
  timestamp();

  FILE *fpt;
  fpt = fopen("task_1.csv", "a");
  fprintf(fpt,"CUDA, %f\n", w_time);
  fclose(fpt);

  return 0;
}

void test(int n_lo, int n_hi, int n_factor)
{
  int n;
  int primes;
  double ctime;

  printf("\n");
  printf("  Call PRIME_NUMBER to count the primes from 1 to N.\n");
  printf("\n");
  printf("         N        Pi          Time\n");
  printf("\n");

  n = n_lo;

  while (n <= n_hi)
  {
    ctime = cpu_time();

    primes = prime_number(n);

    ctime = cpu_time() - ctime;

    printf("  %8d  %8d  %14f\n", n, primes, ctime);
    n = n * n_factor;
  }

  return;
}
